#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "bitmap.h"


#define MAXHISTVAL 0xff
#define NBLOCKS 4 
#define NTHREADS 10

__global__ void histogram(unsigned char *img, int* imgSize, unsigned int *histR, unsigned int *histG, unsigned int *histB) 
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	__shared__ int sharedHistR[MAXHISTVAL + 1];
	__shared__ int sharedHistG[MAXHISTVAL + 1];
	__shared__ int sharedHistB[MAXHISTVAL + 1];

	
	int i = 0;
	for (i; i <= MAXHISTVAL; i++)
	{
		histR[i] = 0;		
		histG[i] = 0;		
		histB[i] = 0;			
		sharedHistR[i] = 0;		
		sharedHistG[i] = 0;		
		sharedHistB[i] = 0;			
	}
	
	__syncthreads();

	/*
	Every specific thread will check image byte starting at index = threadIdx.x + blockIdx.x * blockDim.x
	and then the next ones, each after NBLOCKS * NTHREADS more bytes untill the end is reached.
	*/

	int j = index;
	for(j; j < *imgSize; j += (NBLOCKS*NTHREADS))
	{
		unsigned char pixelClrVal = img[j];
		if(j % 3 == 0)
		{						
			atomicAdd(&sharedHistB[pixelClrVal], 1);
		//	atomicAdd(&histB[pixelClrVal], 1);			
		}
		else if(j % 3 == 1)
		{			
			atomicAdd(&sharedHistG[pixelClrVal], 1);
		//	atomicAdd(&histG[pixelClrVal], 1);			
		}
		else
		{						
			atomicAdd(&sharedHistR[pixelClrVal], 1);
		//	atomicAdd(&histR[pixelClrVal], 1);			
		}	
	}
	__syncthreads();	
	
	if (threadIdx.x == 0)
	{
		int k = 0;
		for (k; k <= MAXHISTVAL; k++)
		{	
			atomicAdd(&histR[k], sharedHistR[k]);
			atomicAdd(&histG[k], sharedHistG[k]);
			atomicAdd(&histB[k], sharedHistB[k]);					
		}	
	}
	__syncthreads();
	
}

/*
* host program
*/
int main(void) {
BITMAPHEADERS bitmapHeaders;
unsigned char* imgData;
int* sizeImgData;
unsigned int *histogramRed;
unsigned int *histogramGreen;
unsigned int *histogramBlue;
int sizeHistogram = (MAXHISTVAL + 1) * sizeof(int);
histogramRed = (unsigned int*) malloc(sizeHistogram);
histogramGreen = (unsigned int*) malloc(sizeHistogram);
histogramBlue = (unsigned int*) malloc(sizeHistogram);

imgData = loadBitmapFile("./img1.bmp", &bitmapHeaders);
if(imgData == NULL)
{
	printf("could not load bitmap file");
	return 1;
}
sizeImgData = &bitmapHeaders.infoHeader.biSizeImage;


 unsigned char* d_imgData;
 int* d_sizeImgData;
 unsigned int *d_histogramRed;
 unsigned int *d_histogramGreen;
 unsigned int *d_histogramBlue;

 int nBlk = NBLOCKS;
 int nThx = NTHREADS;
 //int N = nBlk * nThx;

 // Alloc space for device copies of histogram colors and img data
 hipMalloc((void **)&d_imgData, *sizeImgData);
 hipMalloc((void **)&d_sizeImgData, sizeof(int));
 hipMalloc((void **)&d_histogramRed, sizeHistogram);
 hipMalloc((void **)&d_histogramGreen, sizeHistogram);
 hipMalloc((void **)&d_histogramBlue, sizeHistogram);

 // Copy input data to device
 hipMemcpy(d_imgData, imgData, *sizeImgData, hipMemcpyHostToDevice); 
 hipMemcpy(d_sizeImgData, sizeImgData, sizeof(int), hipMemcpyHostToDevice); 

 // Launch histogram() kernel on GPU with nBlk blocks each with nThx threads
 histogram<<<nBlk,nThx>>>(d_imgData, d_sizeImgData, d_histogramRed, d_histogramGreen, d_histogramBlue);


 // Copy result back to host 
 hipMemcpy(histogramRed, d_histogramRed, sizeHistogram, hipMemcpyDeviceToHost);
 hipMemcpy(histogramGreen, d_histogramGreen, sizeHistogram, hipMemcpyDeviceToHost);
 hipMemcpy(histogramBlue, d_histogramBlue, sizeHistogram, hipMemcpyDeviceToHost);


 printf("CUDA finished.\n Results are saved in the file \n");
 FILE* txtFile = fopen("./cuda_results.txt", "w+");	
 fprintf(txtFile, "Bin value: ");
 int i = 0;
 for(i; i <= MAXHISTVAL; i++)
 {
	fprintf(txtFile, "%u, ", i);
 }

 fprintf(txtFile, "\n Red hist: ");
 int j = 0;
 for(j; j <= MAXHISTVAL; j++)
 {
        fprintf(txtFile, "%u, ", histogramRed[j]);                
 }

 fprintf(txtFile, "\n Green hist:");
 int k = 0;
 for(k; k <= MAXHISTVAL; k++)
 {
        fprintf(txtFile, "%u, ", histogramGreen[k]);
 }

 fprintf(txtFile, "\n Blue hist:");
 int l = 0;
 for(l; l <= MAXHISTVAL; l++)
 {
        fprintf(txtFile, "%u, ", histogramBlue[l]);
 }
 fclose(txtFile);
 
 // Cleanup
 free(imgData); free(histogramRed); free(histogramGreen); free(histogramBlue);
 hipFree(d_imgData); hipFree(d_sizeImgData); hipFree(d_histogramRed); hipFree(d_histogramGreen); hipFree(d_histogramBlue);
 return 0;
}
